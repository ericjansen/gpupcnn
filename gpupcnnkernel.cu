#include "hip/hip_runtime.h"
//////////////////////////////////////////////
// GPU-BASED PULSE-COUPLED NEURAL NETWORK FOR
// CLASSIFYING RETINOPATHY AND MACULAR EDEMA
// developer : ERIC JANSEN
// e-mail : eric[at]jansen[dot]net
// http://www.ericjansen.net
// ONLY WORKING UNDER LINUX
//////////////////////////////////////////////
/*
Copyright (c) 2012, Computer Engineering and Telematics,
Dept. of Electrical Engineering, Institut Teknologi Sepuluh Nopember (ITS)
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:
1. Redistributions of source code must retain the above copyright
   notice, this list of conditions and the following disclaimer.
2. Redistributions in binary form must reproduce the above copyright
   notice, this list of conditions and the following disclaimer in the
   documentation and/or other materials provided with the distribution.
3. All advertising materials mentioning features or use of this software
   must display the following acknowledgement:
   This product includes software developed by Computer Engineering and
   Telematics, Dept. of Electrical Engineering, Institut Teknologi Sepuluh
   Nopember.
4. Neither the name of Institut Teknologi Sepuluh Nopember (ITS) nor the
   names of its contributors may be used to endorse or promote products
   derived from this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY ITS ''AS IS'' AND ANY
EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL [ERIC JANSEN] BE LIABLE FOR ANY
DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include "gpupcnnkernel.cuh"

__global__ void gpuProcessS2(cv::gpu::DevMem2D_<float> S,//const cv::gpu::DevMem2D_<float>& S,
		cv::gpu::DevMem2D_<float> S2)//cv::gpu::DevMem2D_<float>& S2)
{
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;

  if (i > S2.rows || j > S2.cols) return;

  S2.ptr(i)[j] = S.ptr(i)[j]/255.0;
}

__global__ void gpuProcessE(cv::gpu::DevMem2D_<float> E)
{
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;

  if (i > E.rows || j > E.cols) return;

  E.ptr(i)[j] = 2.0;
}

__global__ void gpuPCNN(cv::gpu::DevMem2Df F,
		cv::gpu::DevMem2Df L,
		cv::gpu::DevMem2Df E,
		cv::gpu::DevMem2Df S2,
		cv::gpu::DevMem2D Sum1,
		cv::gpu::PtrStep Y,
		float dAF,float dAL,float dAE,
		float dVF,float dVE,float dB,
    int R,int C)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= R || j >= C) return;

    float dF,dL,dU,dE;

    dF = F.ptr(i)[j];
    dF = exp(-dAF) * F.ptr(i)[j] + S2.ptr(i)[j] + dVF * Sum1.ptr(i)[j];
    F.ptr(i)[j] = dF;

    dL = L.ptr(i)[j];
    dL = exp(-dAL) * dL + L.ptr(i)[j] * Sum1.ptr(i)[j];
    L.ptr(i)[j] = dL;

    dU = F.ptr(i)[j] * (1 + dB * L.ptr(i)[j]);
    dE = E.ptr(i)[j];
    dE = exp(-dAE) * E.ptr(i)[j] + dVE * Y.ptr(i)[j];
    E.ptr(i)[j] = dE;

    if (dU - dE > 0) Y.ptr(i)[j] = 255;
    else Y.ptr(i)[j] = 0;
}

void callGPUProcessS2(cv::gpu::DevMem2D_<float> S,
    cv::gpu::DevMem2D_<float> S2)
{
  dim3 block(16,16);
  dim3 grid((S2.rows+15)/16,(S2.cols+15)/16);
  gpuProcessS2<<<grid,block>>>(S,S2);
}

void callGPUProcessE(cv::gpu::DevMem2D_<float> E)
{
  dim3 block(16,16);
  dim3 grid((E.rows+15)/16,(E.cols+15)/16);
  gpuProcessE<<<grid,block>>>(E);
}

void callGPUKernel(const cv::gpu::DevMem2Df& F,
	const cv::gpu::DevMem2Df& L,
	const cv::gpu::DevMem2Df& E,
	const cv::gpu::DevMem2Df& S2,
  const cv::gpu::DevMem2D& Sum1,
  cv::gpu::PtrStep Y,
//	const cv::gpu::DevMem2D& Y,
	const float& dAF,const float& dAL,const float& dAE,
	const float& dVF,const float& dVE,const float& dB,
  const int& R,const int& C)
{
  cv::gpu::DevMem2D_<float> gF(F),gL(L),gE(E),gS2(S2);
  cv::gpu::DevMem2D_<unsigned char> gSum1(Sum1);//gY(Y);

	dim3 block(16,16);
	dim3 grid((F.cols+block.x-1)/block.x,(F.rows+block.y-1)/block.y);
	gpuPCNN<<<grid,block>>>(gF,gL,gE,gS2,gSum1,Y,
			dAF,dAL,dAE,dVF,dVE,dB,R,C);
}
